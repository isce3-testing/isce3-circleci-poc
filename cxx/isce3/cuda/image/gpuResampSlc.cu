#include "hip/hip_runtime.h"
//-*- coding: utf-8 -*-
//
// Author: Liang Yu
// Copyright: 2018

#include "gpuResampSlc.h"

#include <cmath>

// isce3::core
#include <isce3/core/Constants.h>
#include <isce3/core/Poly2d.h>

#include <isce3/image/Tile.h>
#include <isce3/io/Raster.h>

// isce3::cuda::core
#include <isce3/cuda/core/gpuPoly2d.h>
#include <isce3/cuda/core/gpuLUT1d.h>
#include <isce3/cuda/core/gpuInterpolator.h>

#include <isce3/cuda/except/Error.h>

#include <string>

using isce3::cuda::core::gpuPoly2d;
using isce3::cuda::core::gpuInterpolator;
using isce3::cuda::core::gpuLUT1d;
using isce3::cuda::core::gpuSinc2dInterpolator;

#define THRD_PER_BLOCK 512// Number of threads per block (should always %32==0)

__global__
void transformTile(const thrust::complex<float> *tile,
                   thrust::complex<float> *chip,
                   thrust::complex<float> *imgOut,
                   const float *rgOffTile,
                   const float *azOffTile,
                   const gpuPoly2d rgCarrier,
                   const gpuPoly2d azCarrier,
                   const gpuLUT1d<double> dopplerLUT,
                   gpuSinc2dInterpolator<thrust::complex<float>> interp,
                   bool flatten,
                   int outWidth,
                   int outLength,
                   int inWidth,
                   int inLength,
                   double startingRange,
                   double rangePixelSpacing,
                   double prf,
                   double wavelength,
                   double refStartingRange,
                   double refRangePixelSpacing,
                   double refWavelength,
                   int chipSize,
                   int rowOffset,
                   int rowStart) {

    int iTileOut = blockDim.x * blockIdx.x + threadIdx.x;
    int iChip = iTileOut * chipSize * chipSize;
    int chipHalf = chipSize/2;

    if (iTileOut < outWidth*outLength) {
        int i = iTileOut / outWidth;
        int j = iTileOut % outWidth;
        imgOut[iTileOut] = thrust::complex<float>(0., 0.);

        // Unpack offsets
        const float azOff = azOffTile[iTileOut];
        const float rgOff = rgOffTile[iTileOut];

        // Break into fractional and integer parts
        const int intAz = __float2int_rd(i + azOff + rowStart);
        const int intRg = __float2int_rd(j + rgOff);
        const double fracAz = i + azOff - intAz + rowStart;
        const double fracRg = j + rgOff - intRg;

        // Check bounds again. Use rowOffset to account tiles where tile.rowStart != tile.firstRowImage
        bool intAzInBounds = !((intAz+rowOffset < chipHalf) || (intAz >= (inLength - chipHalf)));
        bool intRgInBounds = !((intRg < chipHalf) || (intRg >= (inWidth - chipHalf)));

        if (intAzInBounds && intRgInBounds) {
            // evaluate Doppler polynomial
            const double rng = startingRange + j * rangePixelSpacing;
            const double dop = dopplerLUT.eval(rng) * 2 * M_PI / prf;

            // Doppler to be added back. Simultaneously evaluate carrier that needs to
            // be added back after interpolation
            double phase = (dop * fracAz)
                + rgCarrier.eval(i + azOff, j + rgOff)
                + azCarrier.eval(i + azOff, j + rgOff);

            // Flatten the carrier phase if requested
            if (flatten) {
                phase += ((4. * (M_PI / wavelength)) *
                    ((startingRange - refStartingRange)
                    + (j * (rangePixelSpacing - refRangePixelSpacing))
                    + (rgOff * rangePixelSpacing))) + ((4.0 * M_PI
                    * (refStartingRange + (j * refRangePixelSpacing)))
                    * ((1.0 / refWavelength) - (1.0 / wavelength)));
            }

            // Modulate by 2*PI
            phase = fmod(phase, 2.0*M_PI);

            // Read data chip without the carrier phases
            for (int ii = 0; ii < chipSize; ++ii) {
                // Row to read from
                const int chipRow = intAz + ii - chipHalf + rowOffset - rowStart;
                // Carrier phase
                const double phase = dop * (ii - 4.0);
                const thrust::complex<float> cval(cos(phase), -sin(phase));
                // Set the data values after removing doppler in azimuth
                for (int jj = 0; jj < chipSize; ++jj) {
                    // Column to read from
                    const int chipCol = intRg + jj - chipHalf;
                    chip[iChip + ii*chipSize+jj] = tile[chipRow*inWidth+chipCol] * cval;
                }
            }

            // Interpolate chip
            const thrust::complex<float> cval = interp.interpolate(
                chipHalf + fracRg, chipHalf + fracAz, &chip[iChip], chipSize, chipSize
            );

            // Add doppler to interpolated value and save
            imgOut[iTileOut] = cval * thrust::complex<float>(cos(phase), sin(phase));
        }
    }
}


// Interpolate tile to perform transformation
void isce3::cuda::image::
gpuTransformTile(isce3::image::Tile<std::complex<float>> & tile,
               isce3::io::Raster & outputSlc,
               isce3::image::Tile<float> & rgOffTile,
               isce3::image::Tile<float> & azOffTile,
               const isce3::core::Poly2d & rgCarrier,
               const isce3::core::Poly2d & azCarrier,
               const isce3::core::LUT1d<double> & dopplerLUT,
               isce3::cuda::core::gpuSinc2dInterpolator<thrust::complex<float>> interp,
               int inWidth, int inLength, double startingRange, double rangePixelSpacing,
               double prf, double wavelength, double refStartingRange,
               double refRangePixelSpacing, double refWavelength,
               bool flatten, int chipSize) {

    // Cache geometry values
    const int outWidth = azOffTile.width();
    const int outLength = azOffTile.length();

    // Allocate valarray for output image block
    std::valarray<std::complex<float>> imgOut(outLength * outWidth);
    // Initialize to zeros
    imgOut = std::complex<float>(0.0, 0.0);

    // declare equivalent objects in device memory
    thrust::complex<float> *d_tile;
    thrust::complex<float> *d_chip;
    thrust::complex<float> *d_imgOut;
    float *d_rgOffTile, *d_azOffTile;
    gpuPoly2d d_rgCarrier(rgCarrier);
    gpuPoly2d d_azCarrier(azCarrier);
    gpuLUT1d<double> d_dopplerLUT(dopplerLUT);

    // determine sizes
    size_t nInPixels = size_t(tile.length()) * tile.width();
    size_t nOutPixels = imgOut.size();
    size_t nOutBytes = nOutPixels * sizeof(thrust::complex<float>);
    size_t nChipBytes = nOutBytes * chipSize * chipSize;

    // allocate equivalent objects in device memory
    checkCudaErrors(hipMalloc(&d_tile, nInPixels*sizeof(thrust::complex<float>)));
    checkCudaErrors(hipMalloc(&d_chip, nChipBytes));
    checkCudaErrors(hipMalloc(&d_imgOut, nOutBytes));
    checkCudaErrors(hipMalloc(&d_azOffTile, nOutPixels*sizeof(float)));
    checkCudaErrors(hipMalloc(&d_rgOffTile, nOutPixels*sizeof(float)));

    // copy objects to device memory
    checkCudaErrors(hipMemcpy(d_tile, &tile[0], nInPixels*sizeof(thrust::complex<float>), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_azOffTile, &azOffTile[0], nOutPixels*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_rgOffTile, &rgOffTile[0], nOutPixels*sizeof(float), hipMemcpyHostToDevice));

    // determine block layout
    dim3 block(THRD_PER_BLOCK);
    dim3 grid((nOutPixels+(THRD_PER_BLOCK-1))/THRD_PER_BLOCK);

    // global call to transform
    transformTile<<<grid, block>>>(d_tile,
                                   d_chip,
                                   d_imgOut,
                                   d_rgOffTile,
                                   d_azOffTile,
                                   d_rgCarrier,
                                   d_azCarrier,
                                   d_dopplerLUT,
                                   interp,
                                   flatten,
                                   outWidth,
                                   outLength,
                                   inWidth,
                                   inLength,
                                   startingRange,
                                   rangePixelSpacing,
                                   prf,
                                   wavelength,
                                   refStartingRange,
                                   refRangePixelSpacing,
                                   refWavelength,
                                   chipSize,
                                   tile.rowStart()-tile.firstImageRow(),// needed to keep az in bounds in subtiles
                                   tile.rowStart());                    // needed to match az components on CPU

    // Check for any kernel errors
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // copy to host memory
    checkCudaErrors(hipMemcpy(&imgOut[0], d_imgOut, nOutBytes, hipMemcpyDeviceToHost));

    // deallocate to device memory
    checkCudaErrors(hipFree(d_tile));
    checkCudaErrors(hipFree(d_chip));
    checkCudaErrors(hipFree(d_imgOut));
    checkCudaErrors(hipFree(d_azOffTile));
    checkCudaErrors(hipFree(d_rgOffTile));

    // Write block of data
    outputSlc.setBlock(imgOut, 0, tile.rowStart(), outWidth, outLength);
}
